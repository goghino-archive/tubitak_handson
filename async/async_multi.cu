#include "hip/hip_runtime.h"
/*
 *
 * Copyright (c) 2014 Juraj Kardos
 *
 * This software is provided 'as-is', without any express or implied warranty.
 * In no event will the authors be held liable for any damages arising 
 * from the use of this software.
 * Permission is granted to anyone to use this software for any purpose, 
 * including commercial applications, and to alter it and redistribute it freely,
 * without any restrictons.
 */

#include <stdio.h>

#include <hip/hip_runtime.h>

#include "nvToolsExt.h"

#define n (16*1014*1024)

__global__ void increment_kernel(int *g_data, int inc_value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx > n/4)
      return;

    //to be more computational intensive, repeat same task multiple times
    for(int i=0; i<30; i++)
        g_data[idx] = g_data[idx] + inc_value;
}

#define checkCudaErrors(cuda_call)  \
{  \
        hipError_t err = (cuda_call);  \
            if (err!=hipSuccess){  \
             printf("\033[31mERROR: %s\n\033[0m", hipGetErrorString(err));  \
             exit(1);  \
            }  \
}


int main(int argc, char *argv[])
{
    int nbytes = n * sizeof(int);
    int value = 26;

    // allocate host memory
    int *a = 0;
    checkCudaErrors(hipHostMalloc((void **)&a, nbytes));
    memset(a, 0, nbytes);

    // allocate device memory
    int *d_a=0;
    checkCudaErrors(hipMalloc((void **)&d_a, nbytes));
    checkCudaErrors(hipMemset(d_a, 255, nbytes));

    // set kernel launch configuration
    dim3 threads = dim3(512, 1);
    dim3 blocks  = dim3(n / threads.x, 1);

    // create cuda event handles
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipDeviceSynchronize());

    //multi GPU related set-up
    int ndevices = 0;
    hipGetDeviceCount(&ndevices);

    int *d_as[ndevices];
    hipEvent_t stop_ev[ndevices];
    // create cuda streams for each device
    hipStream_t stream_multi[4];

    for(int i=0; i<ndevices; i++)
    {
        //select current device
        hipSetDevice(i);

        //create cuda stream for current device
        hipStreamCreate(&stream_multi[i]);

        // allocate device memory
        checkCudaErrors(hipMalloc((void **)&d_as[i], nbytes));
        checkCudaErrors(hipMemset(d_as[i], 255, nbytes/4));

	    //create events
	    hipEventCreate(&stop_ev[i]);
    }

    hipSetDevice(0);
//------------------------------------------------------------------------------

    hipEventRecord(start, 0);
    hipMemcpy(d_a, a, nbytes, hipMemcpyHostToDevice);
    increment_kernel<<<blocks, threads, 0, 0>>>(d_a, value);
    hipEventRecord(stop, 0);

    // have CPU do some work while waiting for stage 1 to finish
    unsigned long int counter=0;

    nvtxRangePushA("CPU Compute");
    while (hipEventQuery(stop) == hipErrorNotReady)
    {
        counter++;
    }
    nvtxRangePop();
    hipMemcpy(a, d_a, nbytes, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();

    float gpu_time_block = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&gpu_time_block, start, stop));
    printf("One big kernel compute time (blocking): %fms\n", gpu_time_block);

//------------------------------------------------------------------------------
    // asynchronously copy data, run kernel and copy back
    hipEventRecord(start, 0);
    hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, 0);
    increment_kernel<<<blocks, threads, 0, 0>>>(d_a, value);
    hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, 0);
    hipEventRecord(stop, 0);

    // have CPU do some work while waiting for stage 1 to finish
    counter=0;

    nvtxRangePushA("CPU Compute");
    while (hipEventQuery(stop) == hipErrorNotReady)
    {
        counter++;
    }
    nvtxRangePop();

    float gpu_time = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));
    printf("One big kernel compute time (async): %fms\n", gpu_time);

//------------------------------------------------------------------------------
// run kernel on partial data multiple times, overlap computation and communication

    // set kernel launch configuration
    threads = dim3(512, 1, 1);
    blocks  = dim3(n / 4 / threads.x, 1, 1);

    // create cuda streams
    hipStream_t stream[4];
    hipStreamCreate(&stream[0]);
    hipStreamCreate(&stream[1]);
    hipStreamCreate(&stream[3]);
    hipStreamCreate(&stream[2]);
    checkCudaErrors(hipDeviceSynchronize());

    int offset = n/4;

    // asynchronously copy data, run kernel and copy back
    hipEventRecord(start, stream[0]);
    hipMemcpyAsync(d_a, a, nbytes/4, hipMemcpyHostToDevice, stream[0]);
    increment_kernel<<<blocks, threads, 0, stream[0]>>>(d_a, value);
    hipMemcpyAsync(a, d_a, nbytes/4, hipMemcpyDeviceToHost, stream[0]);

    hipMemcpyAsync(d_a+offset, a+offset, nbytes/4, hipMemcpyHostToDevice, stream[1]);
    increment_kernel<<<blocks, threads, 0, stream[1]>>>(d_a+offset, value);
    hipMemcpyAsync(a+offset, d_a+offset, nbytes/4, hipMemcpyDeviceToHost, stream[1]);

    hipMemcpyAsync(d_a+2*offset, a+2*offset, nbytes/4, hipMemcpyHostToDevice, stream[2]);
    increment_kernel<<<blocks, threads, 0, stream[2]>>>(d_a+2*offset, value);
    hipMemcpyAsync(a+2*offset, d_a+2*offset, nbytes/4, hipMemcpyDeviceToHost, stream[2]);

    hipMemcpyAsync(d_a+3*offset, a+3*offset, nbytes/4, hipMemcpyHostToDevice, stream[3]);
    increment_kernel<<<blocks, threads, 0, stream[3]>>>(d_a+3*offset, value);
    hipMemcpyAsync(a+3*offset, d_a+3*offset, nbytes/4, hipMemcpyDeviceToHost, stream[3]);
    hipEventRecord(stop, stream[2]);

    // have CPU do some work while waiting for stage 1 to finish
    counter=0;

    nvtxRangePushA("CPU Compute");
    while (hipEventQuery(stop) == hipErrorNotReady)
    {
        counter++;
    }
    nvtxRangePop();

    float gpu_time1 = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&gpu_time1, start, stop));
    printf("Many small kernels compute time: %fms\n", gpu_time1);
    printf("Speedup is %f\n", gpu_time/gpu_time1);

//------------------------------------------------------------------------------

    printf("Found %d CUDA capable devices\n", ndevices);    

    //submit work to GPU devices
    for(int i=0; i<ndevices; i++)
    {    
        hipSetDevice(i);

        hipMemcpyAsync(d_as[i], a+i*offset, nbytes/4, hipMemcpyHostToDevice, stream_multi[i]);
	    increment_kernel<<<blocks, threads, 0, stream_multi[i]>>>(d_as[i], value);
        /*
	    hipError_t errSync  = hipGetLastError();
	    hipError_t errAsync = hipDeviceSynchronize();
	    if (errSync != hipSuccess) 
	        printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	    if (errAsync != hipSuccess)
	        printf("Async kernel %d error: %s\n", i, hipGetErrorString(errAsync));*/
        hipMemcpyAsync(a+i*offset, d_as[i], nbytes/4, hipMemcpyDeviceToHost, stream_multi[i]);
        hipEventRecord(stop_ev[i], stream_multi[i]);
    }

    // have CPU do some work while waiting for stage 1 to finish
    counter=0;

    nvtxRangePushA("CPU Compute");
    while (hipEventQuery(stop_ev[0]) == hipErrorNotReady ||
           hipEventQuery(stop_ev[1]) == hipErrorNotReady ||
           hipEventQuery(stop_ev[2]) == hipErrorNotReady ||
           hipEventQuery(stop_ev[3]) == hipErrorNotReady )  
    {
        counter++;
    }

    /*
    for(int i=0; i<ndevices; i++)
    {
      hipEventSynchronize(stop_ev[i]);  
    }
    */

    //hipDeviceSynchronize();

    nvtxRangePop();

    //checkCudaErrors(hipEventElapsedTime(&gpu_time1, start, stop));
    //printf("Many small kernels compute time: %fms\n", gpu_time1);
    //printf("Speedup is %f\n", gpu_time/gpu_time1);

//------------------------------------------------------------------------------
    // release resources
    hipSetDevice(0);
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipEventDestroy(stop_ev[0]));
    checkCudaErrors(hipEventDestroy(stop_ev[1]));
    checkCudaErrors(hipEventDestroy(stop_ev[2]));
    checkCudaErrors(hipEventDestroy(stop_ev[3]));
    checkCudaErrors(hipHostFree(a));
    checkCudaErrors(hipFree(d_a));
    hipStreamDestroy(stream[0]);
    hipStreamDestroy(stream[1]);
    hipStreamDestroy(stream[2]);
    hipStreamDestroy(stream[3]);

    //free memory
    for(int i=0; i<ndevices; i++)
    {
        hipSetDevice(i);
        hipFree(d_as[i]);
        hipStreamDestroy(stream_multi[i]);
    }

    // flush all profile data
    hipDeviceReset();

}
